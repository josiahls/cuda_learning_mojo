#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <stdio.h>

// From pg 40
#define CHECK(call) \
{ \
	const hipError_t error = call; \
	if (error != hipSuccess) \
	{ \
		printf("Error: %s:%d, ", __FILE__, __LINE__); \
		printf("cuda:%d, reason: %s\n", error, hipGetErrorString(error)); \
		exit(1); \
	} \
}

void sumArraysOnHost(float *A, float *B, float *C, const int N) {
    for (int idx = 0; idx < N; idx++) {
        C[idx] = A[idx] + B[idx];
    }
}


__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
    // NOTE: Doing this will only work if there is only 1 block!
    // int i = threadIdx.x;
    // NOTE: This scales to multiple blocks!
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) C[i] = A[i] + B[i];
}


void initialData(float *ip, int size) {
    // Generate different seed for random number
    time_t t;
    srand((unsigned int) time(&t));
    for (int i = 0; i < size; i++) {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }
}

int main() {
    int nElem = 1024;
    size_t nBytes = nElem * sizeof(float);

    // Initialize the host values and assign the starting data to them.
    float *h_A = (float *)malloc(nBytes);
    float *h_B = (float *)malloc(nBytes);
    float *h_C = (float *)malloc(nBytes);
    float *h_C_check = (float *)malloc(nBytes);
    initialData(h_A, nElem);
    initialData(h_B, nElem);

    // Initialize the device / gpu values
    float *d_A, *d_B, *d_C;
    hipMalloc((float **)&d_A, nBytes);
    hipMalloc((float **)&d_B, nBytes);
    hipMalloc((float **)&d_C, nBytes);

    // Copy the host data to the device allocated memory
    // Important note:
    //  - dst <- src. The kind tells us which devices / hosts we are moving to.
    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);

    // dim3 block(nElem);
    // dim3 grid(nElem / block.x);
    dim3 block(1);
    dim3 grid(nElem);

    // sumArraysOnHost(h_A, h_B, h_C, nElem);
    sumArraysOnGPU<<<grid, block>>>(d_A, d_B, d_C, nElem);

    // Sum the host data to check the result
    sumArraysOnHost(h_A, h_B, h_C_check, nElem);

    // Note cudaMemCpy is always device, host
    CHECK(hipMemcpy(h_C, d_C,  nBytes, hipMemcpyDeviceToHost));

    for (int i = 0; i < nElem; i++) {
        printf("%f + %f = %f vs %f\n", h_A[i], h_B[i], h_C[i], h_C_check[i]);
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}