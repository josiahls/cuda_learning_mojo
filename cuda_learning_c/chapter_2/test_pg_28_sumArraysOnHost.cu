#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <stdio.h>

// From pg 40
#define CHECK(call) \
{ \
	const hipError_t error = call; \
	if (error != hipSuccess) \
	{ \
		printf("Error: %s:%d, ", __FILE__, __LINE__); \
		printf("cuda:%d, reason: %s\n", error, hipGetErrorString(error)); \
		exit(1); \
	} \
}

void sumArraysOnHost(float *A, float *B, float *C, const int N) {
    for (int idx = 0; idx < N; idx++) {
        C[idx] = A[idx] + B[idx];
    }
}


__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
    int i = threadIdx.x;

    if (i < N) C[i] = A[i] + B[i];
}


void initialData(float *ip, int size) {
    // Generate different seed for random number
    time_t t;
    srand((unsigned int) time(&t));
    for (int i = 0; i < size; i++) {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }
}

int main() {
    int nElem = 1024;
    size_t nBytes = nElem * sizeof(float);

    // Initialize the host values and assign the starting data to them.
    float *h_A = (float *)malloc(nBytes);
    float *h_B = (float *)malloc(nBytes);
    float *h_C = (float *)malloc(nBytes);
    initialData(h_A, nElem);
    initialData(h_B, nElem);

    // Initialize the device / gpu values
    float *d_A, *d_B, *d_C;
    hipMalloc((float **)&d_A, nBytes);
    hipMalloc((float **)&d_B, nBytes);
    hipMalloc((float **)&d_C, nBytes);

    // Copy the host data to the device allocated memory
    // Important note:
    //  - dst <- src. The kind tells us which devices / hosts we are moving to.
    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);

    // sumArraysOnHost(h_A, h_B, h_C, nElem);
    sumArraysOnGPU<<<1, nElem>>>(d_A, d_B, d_C, nElem);

    // Note cudaMemCpy is always device, host
    CHECK(hipMemcpy(h_C, d_C,  nBytes, hipMemcpyDeviceToHost));

    for (int i = 0; i < nElem; i++) {
        printf("%f + %f = %f\n", h_A[i], h_B[i], h_C[i]);
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}